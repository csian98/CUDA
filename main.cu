/**
 * @file		main.cu
 * @brief		
 * @author		Jeong Hoon (Sian) Choi
 * @version 	1.0.0
 * @date		2024-04-03
 */

/* Copyright (C)
 * 2024 - Jeong Hoon (Sian) Choi
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * as published by the Free Software Foundation; either version 2
 * of the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the Free Software
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include "sample.cuh"

#if _TARGET_OS == OS_WINDOWS

#elif _TARGET_OS == OS_LINUX

// void custom_terminate_fnct(void) {
//	exit(1);
// }

__host__ int main(int argc, char* argv[]) {
// 	std::terminate_handler default_terminate =
//	std::set_terminate(&custom_terminate_fnct);
	void *d_ptr, *h_ptr;
	size_t allocate = 4096;

	h_ptr = malloc(sizeof(std::byte)*allocate);
	memset(h_ptr, 0, sizeof(std::byte)*allocate);

	// memory processing in host
	
    if (auto error = hipMalloc(&d_ptr, sizeof(std::byte)*allocate) != hipSuccess) {
		std::cerr << hipGetErrorName(error) << std::endl;
		exit(1);
	}
	
	if (auto error = hipMemset(&d_ptr, 0, sizeof(std::byte)*allocate) != hipSuccess) {
		std::cerr << hipGetErrorName(error) << std::endl;
		exit(1);
	}

	if (auto error = hipMemcpy(d_ptr, h_ptr, sizeof(std::byte)*allocate) != hipSuccess) {
		std::cerr << hipGetErrorName(error) << std::endl;
		exit(1);
	}

//	kernel<<<1, 10>>>();

    if (auto error = hipDeviceSynchronize() != hipSuccess) {
		std::cerr << hipGetErrorName(error) << std::endl;
		exit(1);
	}

	if (auto error = hipMemcpy(h_ptr, d_ptr, sizeof(std::byte)*allocate, hipMemcpyDeviceToHost) != hipSuccess) {
		std::cerr << hipGetErrorName(error) << std::endl;
		exit(1);
	}
	
	if (auto error = hipFree(d_ptr) != hipSuccess) {
		std::cerr << hipGetErrorName(error) << std::endl;
		exit(1);
	}
	free(h_ptr);

	return 0;
}

#endif // OS dependency
