#include <hip/hip_runtime.h>



#include <stdio.h>
#include <stdlib.h>

__host__ void checkDeviceMemory(void) {
	size_t free, total;
	hipMemGetInfo(&free, &total);
	printf("Device memory (free/total) = %lld/%lld bytes\n", free, total);
}

__host__ int main(void) {
	int* dDataPtr;
	hipError_t errorCode;

	checkDeviceMemory();
	errorCode=hipMalloc(&dDataPtr, sizeof(int)*1024*1024*1024);
	printf("hipMalloc - %s\n", hipGetErrorName(errorCode));
	checkDeviceMemory();

	errorCode==hipMemset(dDataPtr, 0, sizeof(int)*1024*1024*1024);
	printf("hipMemset - %s\n", hipGetErrorName(errorCode));
	
	errorCode=hipFree(dDataPtr);
	printf("hipFree - %s\n", hipGetErrorName(errorCode));
	checkDeviceMemory();
}
